#include <hip/hip_runtime.h.hpp>
#include <work1/kernels/kernel_vector_add.cuh>
#include <work1/vector.cuh>

#define EIGEN_NO_CUDA

#include <Eigen/Dense>
#include <benchmark/benchmark.h>
#include <cuda_timer.hpp>

static void BM_EigenVectorAddCPU(benchmark::State& state) {
  auto len = state.range(0);

  Eigen::VectorXf a = Eigen::VectorXf(len);
  Eigen::VectorXf b = Eigen::VectorXf(len);
  Eigen::VectorXf result(len);

  for (auto _ : state) {
    result = a + b;  // lazy RHS
    benchmark::DoNotOptimize(result.data());
    benchmark::ClobberMemory();
  }
}

static void BM_OurVectorAddGPU(benchmark::State& state) {
  auto size = state.range(0);

  auto a = hsys::Vector<float>(size);
  auto b = hsys::Vector<float>(size);
  auto c = hsys::Vector<float>(size);

  for (auto _ : state) {
    float elapsed_time = 0;

    {
      CUDATimer timer(elapsed_time);
      hsys::kernel_vector_add<<<hip/hip_runtime.h::cover(size, 128), 128>>>(
          c.accessor(), a.accessor(), b.accessor());
    }

    benchmark::DoNotOptimize(elapsed_time);
    benchmark::ClobberMemory();

    state.SetIterationTime(elapsed_time);
  }
}

constexpr const int multiplier = 8;
constexpr const auto range = std::make_pair(8, 1 << 26);
constexpr const auto unit = benchmark::kMillisecond;

BENCHMARK(BM_EigenVectorAddCPU)
    ->Name("Eigen Vector Addition (CPU)")
    ->RangeMultiplier(multiplier)
    ->Ranges({range})
    ->Unit(unit)
    ->UseRealTime()
    ->MeasureProcessCPUTime();

BENCHMARK(BM_OurVectorAddGPU)
    ->Name("CUDA Vector Addition (GPU)")
    ->RangeMultiplier(multiplier)
    ->Ranges({range})
    ->Unit(unit)
    ->UseManualTime();

BENCHMARK_MAIN();  // NOLINT
